#include "hip/hip_runtime.h"
//Color based segmentation 

#include "utils.h"
#include <stdio.h>



//KERNEL PARA CONVERSIÓN A HSV
__global__
void rgba_2_hsv(const uchar4* const rgbaImage,
                       uchar3* const hsvImage,
                       int numRows, int numCols)
{ 

  int y = threadIdx.y+ blockIdx.y* blockDim.y;   //globalIdx = (blockIdx * threadsPerBlock) + threadId

  int x = threadIdx.x+ blockIdx.x* blockDim.x;
float rgbaMAX=0;
float rgbaMIN=0;

//prevents accessing out of bounds
if (y < numCols && x < numRows) 
  {
  	int index = numRows*y +x;    ///numCols
        /// printf("index = %d\n",index);

//CONVERT 8 B TO FLOAT 
float R=rgbaImage[index].x*(1.0/255.0), G=rgbaImage[index].y*(1.0/255.0), B=rgbaImage[index].z*(1.0/255.0);

//FIND MAX AND MIN VALUES FOR THE RGB STRUCT

if(B > G){
	if(B > R){
	     rgbaMAX= B; //B CHANNEL MAX VAlUE
	    
	       if(G > R){
	        	rgbaMIN= R;}
	       else{rgbaMIN= G;}
	}else{rgbaMAX=R;
		rgbaMIN=G;}
  }else{
	if(G > R){
	      rgbaMAX= G;
	      if(B > R){
	      rgbaMIN= R;}
	      else{rgbaMIN= B;}
	}else{rgbaMAX= R;
	      rgbaMIN= B;}
  }

unsigned char V = rgbaMAX*(255); /// V=MAX(R,G,B)
unsigned char S=0;
unsigned char H=0;
float Sp=0, Hp=0;

//Saturation
if(V != 0)
  {Sp=((rgbaMAX-rgbaMIN)/rgbaMAX); } ///  S= (V-min(R,G,B)) / V }
S=Sp*(255);

//hue ineficiente
if(V==R*255){
   if(G>=B){
     Hp=(60*(G-B))/(rgbaMAX-rgbaMIN);}
 else{    
	Hp=((60*(G-B))/(rgbaMAX-rgbaMIN) )+360;}
}
if(V==G*255){Hp=((60*(B-R))/(rgbaMAX-rgbaMIN))+120;}
if(V==B*255 && V!=G*255 && V!=R*255){ Hp=((60*(R-G))/(rgbaMAX-rgbaMIN))+240;}
H=Hp*(0.5);

if(H==0){H=1;}
 

hsvImage[index].x= H;
hsvImage[index].y= S;
hsvImage[index].z= V;

}
}


__global__ void threshold_kernel(const uchar3* hsvImage,
 	     	  	    	  unsigned char* thresImage,				  
 				  int numRows, int numCols){
int Hmin=90, Smin=120, Vmin=100;
// int Hmin=100, Smin=100, Vmin=110;
//int Hmin=0, Smin=0, Vmin=0;
int Hmax=170, Smax=250, Vmax=250;

  int y = threadIdx.y+ blockIdx.y* blockDim.y;   //globalIdx = (blockIdx * threadsPerBlock) + threadId

   int x = threadIdx.x+ blockIdx.x* blockDim.x;

if (y < numCols && x < numRows) 
  {
   	int index = numRows*y +x;


unsigned char H=hsvImage[index].x;
unsigned char S=hsvImage[index].y;
unsigned char V=hsvImage[index].z;

if(H>Hmin && H<Hmax && S>Smin && S<Smax && V>Vmin && V<Vmax){
  thresImage[index]=255;  

 }else{thresImage[index]=0; }


}

 }




//KERNEL FOR EROSION

__global__ void erode_kernel(unsigned char * thresImage,
 	     	  	    	  unsigned char* erodedImage, //unsigned char* dilatedImage,
 				  int numRows, int numCols){

int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;
int menor=255;

//extern  __shared__ unsigned char sh_thresImage[];

 if( y >= numCols || x>= numRows){
     return;}

    	int index = numCols*x +y;

//sh_thresImage[index]=thresImage[index];
//__syncthreads();

//Kernel with 2D VON NEUMMAN stencil pattern
//vertical values for the operator
//max and min to avoid accesing  out of bounds. a la posiciṕnen el grid se le suma una cantidad de posiciones igual al tamaño del kernel, después se desplaza por la mitad de su tamaño}

// int kernelSize = 4;
// for(int i=0; i<kernelSize;i++){
// 	int offsetX= min(max(x + i -kernelSize/2,0), numRows -1);
// 	int temp= thresImage[offsetX*numCols +y];
// 	if(temp < menor){
// 		 menor=temp;
// 		 }}

// for(int i=0; i<kernelSize;i++){    //   horizontal
// 	int offsetY= min(max(y + i -kernelSize/2,0), numCols -1);
// 	int temp= thresImage[x*numCols + offsetY];
// 	if(temp< menor){
// 		 menor=temp;
// 		 }}
 
//Kernel rectangular stencil pattern
int kernelWidth =4;
int kernelHeight =4;

for(int i=0; i<kernelWidth;i++){
 	int offsetY= min(max(y + i -kernelWidth/2,0), numCols -1);
      for(int j=0; j<kernelHeight;j++){
	int offsetX= min(max(x + i -kernelHeight/2,0), numRows -1);
	
 	int temp= thresImage[offsetX*numCols + offsetY];
	if(temp< menor){
 		 menor=temp;
 		 }

             }
	}


 erodedImage[index]=menor;
}



////KERNEL DILATACIÓN
__global__ void dilate_kernel(unsigned char * erodedImage,
 	     	  	    	  unsigned char* dilatedImage,
 				  int numRows, int numCols){

int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;
int mayor=0;

if( y >= numCols || x>= numRows){
    return;}

int index = numCols*x +y;   


//Kernel with 2D VON NEUMMAN stencil pattern

//vertical values for the operator
//max and min to avoid accesing  out of bounds. a la posiciṕnen el grid se le suma una cantidad de posiciones igual al tamaño del kernel, después se desplaza por la mitad de su tamaño}

int kernelSize = 4;
for(int i=0; i<kernelSize;i++){
	int offsetX= min(max(x + i -kernelSize/2,0), numRows -1);
	int temp= erodedImage[offsetX*numCols + y];
	if(temp > mayor){
		 mayor=temp;
		 }}


for(int i=0; i<kernelSize;i++){   //horizontal values
	int offsetY= min(max(y + i -kernelSize/2,0), numCols -1);
	int temp= erodedImage[x*numCols + offsetY ];
	if(temp > mayor){
		 mayor=temp;
		 }}
		 
//Kernel rectangular stencil pattern
int kernelWidth =4;
int kernelHeight =4;

for(int i=0; i<kernelWidth;i++){
 	int offsetY= min(max(y + i -kernelWidth/2,0), numCols -1);
      for(int j=0; j<kernelHeight;j++){
	int offsetX= min(max(x + i -kernelHeight/2,0), numRows -1);
	
 	int temp= erodedImage[offsetX*numCols + offsetY];
	if(temp > mayor){
 		 mayor=temp;
 		 }}}



dilatedImage[index]=mayor;
}


__global__ void window_hgw_kernel(unsigned char * thresImage,
 	     	  	    	  unsigned char* windowImage, 
 				  int numRows, int numCols){

		int p=5;
		int apron = 2;
		int cont=0;
		int i=0;
		int indicehgw=0 ;
		int indice =0;
	
		
		for(int x=0;x<numRows;x++){
		  for(int y=0;y<=numCols;y++){  ///last bug
		    indice=x*numCols + y;//x*numRows + y ;

		    //h_erohgw_o[indice]=h_thresImage[indice];

                    if(cont==0){ //agregar apron izquierdo; inicia una nueva ventana
		      
		      for(int c=0; c<apron; c++){
		    	windowImage[indicehgw]=0;             //0:dilat|| 255||ero
		    	// printf("%u \n", h_ero_hgw[indicehgw]);
		        indicehgw++;
		        
		        }
		      
		    }

		         if (cont>=p){ //agregar apron derecho; finalizar ventana
		           
		    	   for(int c=0; c<apron; c++){
			    
		    	     windowImage[indicehgw]=0; 
		    	     // printf("%u \n", h_ero_hgw[indicehgw]);
		             indicehgw++;
			     
		    	     }
		              cont = 0; //reiniciar contador de ventana
			      y=y-1;  //regresar 1 posición en la fila
			       
		         }else{
			 
		    	    windowImage[indicehgw]=thresImage[indice];
		    	    // printf("%u  ", h_ero_hgw[indicehgw]);
		    	    // printf("%u \n", h_thresImage[indice]);
		           cont++;
		           indicehgw++;
			 
			 }
			 
			       
		    
		  }}




}


__global__ void ero_hgw_kernel(unsigned char * windowImage,
 	     	  	    	  unsigned char* erohgwImage, 
 				  int numRows, int numCols){

}





void color_seg(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            uchar3 * const d_hsvImage, unsigned char * d_thresImage,
			    unsigned char * d_erodedImage, unsigned char * d_dilatedImage,
			    unsigned char * d_window_hgw, unsigned char * d_ero_hgw,
			    size_t numRows, size_t numCols)
{


  
  int   blockWidth = 8;   // (dimensionX / gridbloqueenX) = threadsporbloqueenX

    const dim3 blockSize(blockWidth, blockWidth, 1);
   int   blocksX = (numRows/blockWidth)+1;       // +1 por truncamiento
   int   blocksY = numCols/blockWidth +1; 
   const dim3 gridSize( blocksX, blocksY, 1);  
   int pixel=numRows*numCols;

///////////////

  rgba_2_hsv<<<gridSize, blockSize>>>(d_rgbaImage, d_hsvImage, numRows, numCols); 
 // hipDeviceSynchronize();
  threshold_kernel<<<gridSize, blockSize>>>(d_hsvImage, d_thresImage, numRows, numCols);
  window_hgw_kernel<<<1,1>>>(d_thresImage,d_window_hgw, numRows, numCols);
 // hipDeviceSynchronize();

ero_hgw_kernel<<<gridSize, blockSize>>>(d_window_hgw, d_ero_hgw, numRows, numCols);

 // erode_kernel<<<gridSize,blockSize>>>(d_thresImage, d_erodedImage, numRows, numCols);
 // // hipDeviceSynchronize();
 //  dilate_kernel<<<gridSize,blockSize>>>(d_erodedImage, d_dilatedImage,numRows, numCols);
  
  //checkCudaErrors(hipGetLastError());
}


