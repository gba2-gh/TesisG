#include "hip/hip_runtime.h"
//Color based segmentation 

#include "utils.h"
#include <stdio.h>



//KERNEL PARA CONVERSIÓN A HSV
__global__
void rgba_2_hsv(const uchar4* const rgbaImage,
                       uchar3* const hsvImage,
                       int numRows, int numCols)
{ 

  int y = threadIdx.y+ blockIdx.y* blockDim.y;   //globalIdx = (blockIdx * threadsPerBlock) + threadId

  int x = threadIdx.x+ blockIdx.x* blockDim.x;
float rgbaMAX=0;
float rgbaMIN=0;

//prevents accessing out of bounds
if (y < numCols && x < numRows) 
  {
  	int index = numRows*y +x;    ///numCols
        /// printf("index = %d\n",index);

//CONVERT 8 B TO FLOAT 
float R=rgbaImage[index].x*(1.0/255.0), G=rgbaImage[index].y*(1.0/255.0), B=rgbaImage[index].z*(1.0/255.0);

//FIND MAX AND MIN VALUES FOR THE RGB STRUCT

if(B > G){
	if(B > R){
	     rgbaMAX= B; //B CHANNEL MAX VAlUE
	    
	       if(G > R){
	        	rgbaMIN= R;}
	       else{rgbaMIN= G;}
	}else{rgbaMAX=R;
		rgbaMIN=G;}
  }else{
	if(G > R){
	      rgbaMAX= G;
	      if(B > R){
	      rgbaMIN= R;}
	      else{rgbaMIN= B;}
	}else{rgbaMAX= R;
	      rgbaMIN= B;}
  }

unsigned char V = rgbaMAX*(255); /// V=MAX(R,G,B)
unsigned char S=0;
unsigned char H=0;
float Sp=0, Hp=0;

//Saturation
if(V != 0)
  {Sp=((rgbaMAX-rgbaMIN)/rgbaMAX); } ///  S= (V-min(R,G,B)) / V }
S=Sp*(255);

//hue ineficiente
if(V==R*255){
   if(G>=B){
     Hp=(60*(G-B))/(rgbaMAX-rgbaMIN);}
 else{    
	Hp=((60*(G-B))/(rgbaMAX-rgbaMIN) )+360;}
}
if(V==G*255){Hp=((60*(B-R))/(rgbaMAX-rgbaMIN))+120;}
if(V==B*255 && V!=G*255 && V!=R*255){ Hp=((60*(R-G))/(rgbaMAX-rgbaMIN))+240;}
H=Hp*(0.5);

if(H==0){H=1;}
 

hsvImage[index].x= H;
hsvImage[index].y= S;
hsvImage[index].z= V;

}
}


__global__ void threshold_kernel(const uchar3* hsvImage,
 	     	  	    	  unsigned char* thresImage,				  
 				  int numRows, int numCols){
int Hmin=90, Smin=120, Vmin=100;
// int Hmin=100, Smin=100, Vmin=110;
//int Hmin=0, Smin=0, Vmin=0;
int Hmax=170, Smax=250, Vmax=250;

  int y = threadIdx.y+ blockIdx.y* blockDim.y;   //globalIdx = (blockIdx * threadsPerBlock) + threadId

   int x = threadIdx.x+ blockIdx.x* blockDim.x;

if (y < numCols && x < numRows) 
  {
   	int index = numRows*y +x;


unsigned char H=hsvImage[index].x;
unsigned char S=hsvImage[index].y;
unsigned char V=hsvImage[index].z;

if(H>Hmin && H<Hmax && S>Smin && S<Smax && V>Vmin && V<Vmax){
  thresImage[index]=255;  

 }else{thresImage[index]=0; }


}

 }




//KERNEL FOR EROSION

__global__ void erode_kernel(unsigned char * thresImage,
 	     	  	    	  unsigned char* erodedImage, //unsigned char* dilatedImage,
 				  int numRows, int numCols){

int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;
int menor=255;

//extern  __shared__ unsigned char sh_thresImage[];

 if( y >= numCols || x>= numRows){
     return;}

    	int index = numCols*x +y;

//sh_thresImage[index]=thresImage[index];
//__syncthreads();

//Kernel with 2D VON NEUMMAN stencil pattern
//vertical values for the operator
//max and min to avoid accesing  out of bounds. a la posiciṕnen el grid se le suma una cantidad de posiciones igual al tamaño del kernel, después se desplaza por la mitad de su tamaño}

// int kernelSize = 4;
// for(int i=0; i<kernelSize;i++){
// 	int offsetX= min(max(x + i -kernelSize/2,0), numRows -1);
// 	int temp= thresImage[offsetX*numCols +y];
// 	if(temp < menor){
// 		 menor=temp;
// 		 }}

// for(int i=0; i<kernelSize;i++){    //   horizontal
// 	int offsetY= min(max(y + i -kernelSize/2,0), numCols -1);
// 	int temp= thresImage[x*numCols + offsetY];
// 	if(temp< menor){
// 		 menor=temp;
// 		 }}
 
//Kernel rectangular stencil pattern
int kernelWidth =4;
int kernelHeight =4;

for(int i=0; i<kernelWidth;i++){
 	int offsetY= min(max(y + i -kernelWidth/2,0), numCols -1);
      for(int j=0; j<kernelHeight;j++){
	int offsetX= min(max(x + i -kernelHeight/2,0), numRows -1);
	
 	int temp= thresImage[offsetX*numCols + offsetY];
	if(temp< menor){
 		 menor=temp;
 		 }

             }
	}


 erodedImage[index]=menor;
}



////KERNEL DILATACIÓN
__global__ void dilate_kernel(unsigned char * erodedImage,
 	     	  	    	  unsigned char* dilatedImage,
 				  int numRows, int numCols){

int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;
int mayor=0;

if( y >= numCols || x>= numRows){
    return;}

int index = numCols*x +y;   


//Kernel with 2D VON NEUMMAN stencil pattern

//vertical values for the operator
//max and min to avoid accesing  out of bounds. a la posiciṕnen el grid se le suma una cantidad de posiciones igual al tamaño del kernel, después se desplaza por la mitad de su tamaño}

int kernelSize = 4;
for(int i=0; i<kernelSize;i++){
	int offsetX= min(max(x + i -kernelSize/2,0), numRows -1);
	int temp= erodedImage[offsetX*numCols + y];
	if(temp > mayor){
		 mayor=temp;
		 }}


for(int i=0; i<kernelSize;i++){   //horizontal values
	int offsetY= min(max(y + i -kernelSize/2,0), numCols -1);
	int temp= erodedImage[x*numCols + offsetY ];
	if(temp > mayor){
		 mayor=temp;
		 }}
		 
//Kernel rectangular stencil pattern
int kernelWidth =4;
int kernelHeight =4;

for(int i=0; i<kernelWidth;i++){
 	int offsetY= min(max(y + i -kernelWidth/2,0), numCols -1);
      for(int j=0; j<kernelHeight;j++){
	int offsetX= min(max(x + i -kernelHeight/2,0), numRows -1);
	
 	int temp= erodedImage[offsetX*numCols + offsetY];
	if(temp > mayor){
 		 mayor=temp;
 		 }}}



dilatedImage[index]=mayor;
}



//EROSIÓNN///////////////////////////////////////////////////////////////////
__global__ void ero_hgw_kernel_hor(unsigned char * thresImage,
 	     	  	    	  unsigned char* suffix, 
				  unsigned char * prefix,
 				  int rows, int cols){


int offset =0;
int p=7;
unsigned char w[10]={0} ;
int c=0;
int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;

//printf("y= %i\t", y);
if( y >= cols/p || x>= rows){
    return;}

int index = cols*x +y; 

//HORIZONTAL


 //arreglo s
 
   offset= p*y  + x*cols;
   
   suffix[offset + p -1] = thresImage[offset +p -1]; // agregar primer dato
   //w[c]=thresImage[offset +p -1];
   //printf("s=%u ",h_erohgw[offset+p-1]);
     for(int j=offset + p -2; j>=offset; j--){
       suffix[j]=min(thresImage[j], suffix[j+1]);
       //c++;
       //w[c]= thresImage[j];
         //printf("s=%u ",suffix[j]);
     }

     prefix[offset] = thresImage[offset] ;// agregar primer dato
     //printf("r=%u ",h_dilhgw[offset]);
     for(int j=offset + 1; j<=(offset + (p-1)); j++){
     
       prefix[j]=min(thresImage[j], prefix[j-1]);
      // c++;
       //w[c]=thresImage[j];
       // printf("r=%u ",prefix[j]);
       
    }
    
   // printf("\n");
   // printf("y= %i\t", y);
     // for(int i=0; i<= 2*(p-1); i++){
     //   printf("w=%u ",w[i]); 
     // }

     //c=0;

 
}


__global__ void ero_hgw_kernel_ver(unsigned char * thresImage,
 	     	  	    	  unsigned char* suffix, 
				  unsigned char * prefix,
 				  int rows, int cols){


int offset =0;
int p=7;
int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;
 if( x>= cols || y>=rows/p){
      return;}

//printf("x=%i\t",x);
//   //VERTICAL
 //arreglo s
 
   offset=(p*y)*cols + x; 
   suffix[offset + (p -1)*cols] = thresImage[offset + (p -1)*cols]; // agregar primer dato
   
   for(int j=offset + (p -2)*cols; j>=offset; j=j-cols){
       suffix[j]=min(thresImage[j], suffix[j+cols]);
     }

   prefix[offset] = thresImage[offset] ;// agregar primer dato
   for(int j=offset + cols; j<=(offset + (p-1)*cols); j=j+cols){
     
       prefix[j]=min(thresImage[j], prefix[j-cols]);
       
    }

 
}
/////////////////////////////////////////////////////EROSIÓN/////////////////////////////////////



/////DILATACIÓN///////////////////////////////////////////////////////////////////////////////


__global__ void dil_hgw_kernel_hor(unsigned char * erohgw,
 	     	  	    	  unsigned char* suffix, 
				  unsigned char * prefix,
 				  int rows, int cols){


int offset =0;
int p=7;

int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;

if( y >= cols/p || x>= rows){
    return;}

///////HORIZONTAL////////

 //arreglo s
 
   offset= p*y  + x*cols;
   
   suffix[offset + p -1] = erohgw[offset +p -1]; // agregar primer dato
     for(int j=offset + p -2; j>=offset; j--){
       suffix[j]=max(erohgw[j], suffix[j+1]);
     }

     prefix[offset] = erohgw[offset] ;// agregar primer dato

   for(int j=offset + 1; j<=(offset + (p-1)); j++){    
       prefix[j]=max(erohgw[j], prefix[j-1]);   
    }


 
}


__global__ void dil_hgw_kernel_ver(unsigned char * thresImage,
 	     	  	    	  unsigned char* suffix, 
				  unsigned char * prefix,
 				  int rows, int cols){


int offset =0;
int p=7;
int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;
 if( x>= cols || y>=rows/p){
      return;}

//printf("x=%i\t",x);
//   //VERTICAL
 //arreglo s
   offset=(p*y)*cols + x; 
   suffix[offset + (p -1)*cols] = thresImage[offset + (p -1)*cols]; // agregar primer dato
   
   for(int j=offset + (p -2)*cols; j>=offset; j=j-cols){
       suffix[j]=max(thresImage[j], suffix[j+cols]);
     }

   prefix[offset] = thresImage[offset] ;// agregar primer dato
   for(int j=offset + cols; j<=(offset + (p-1)*cols); j=j+cols){
     
       prefix[j]=max(thresImage[j], prefix[j-cols]);
       
    }


}

///////////////////////////////////////////////////////DILATACIÓN//////////////////////////


__global__ void ero_result_kernel(unsigned char *suffix, unsigned char *prefix, unsigned char *  erohgw, int rows, int cols){

int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;

if( y >= cols || x>= rows){
    return;}
    
//printf("y= %i\t", y);
int index = cols*x +y; 
     erohgw[index]=min(suffix[index],prefix[index]); //result 
}


__global__ void dil_result_kernel(unsigned char *suffix, unsigned char *prefix, unsigned char *  erohgw, int rows, int cols){

int x = blockIdx.x * blockDim.x +  threadIdx.x;
int y = blockIdx.y *blockDim.y + threadIdx.y;

if( y >= cols || x>= rows){
    return;}

int index = cols*x +y; 
     erohgw[index]=max(suffix[index],prefix[index]); //result 
}




void color_seg(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            uchar3 * const d_hsvImage, unsigned char * d_thresImage,
			    unsigned char * d_erodedImage, unsigned char * d_dilatedImage,
			    unsigned char * d_erohgw,unsigned char * d_dilhgw, unsigned char * d_suffix, unsigned char *d_prefix,
			    size_t numRows, size_t numCols)
{


  int p=5;
  int   blockWidth = 8;   // (dimensionX / gridbloqueenX) = threadsporbloqueenX

   const dim3 blockSize(blockWidth, blockWidth, 1);
   int   blocksX = (numRows/blockWidth)+1;       // +1 por truncamiento
   int   blocksY = (numCols/blockWidth) +1 ; 
   const dim3 gridSize( blocksX, blocksY , 1);  
  

///////////////

  rgba_2_hsv<<<gridSize, blockSize>>>(d_rgbaImage, d_hsvImage, numRows, numCols); 
 // hipDeviceSynchronize();
  threshold_kernel<<<gridSize, blockSize>>>(d_hsvImage, d_thresImage, numRows, numCols);

 // hipDeviceSynchronize();

ero_hgw_kernel_hor<<<gridSize,blockSize,1>>>(d_thresImage, d_suffix, d_prefix,  numRows, numCols);
ero_result_kernel<<<gridSize, blockSize,1>>>(d_suffix, d_prefix, d_erohgw, numRows, numCols);
ero_hgw_kernel_ver<<<gridSize,blockSize,1>>>(d_erohgw, d_suffix, d_prefix,  numRows, numCols);
ero_result_kernel<<<gridSize, blockSize,1>>>(d_suffix, d_prefix, d_erohgw, numRows, numCols);

hipDeviceSynchronize();

dil_hgw_kernel_hor<<<gridSize, blockSize,1>>>(d_erohgw, d_suffix, d_prefix,  numRows, numCols);
dil_result_kernel<<<gridSize, blockSize,1>>>(d_suffix, d_prefix, d_dilhgw, numRows, numCols);
dil_hgw_kernel_ver<<<gridSize, blockSize,1>>>(d_dilhgw, d_suffix, d_prefix,  numRows, numCols);
dil_result_kernel<<<gridSize, blockSize,1>>>(d_suffix, d_prefix, d_dilhgw, numRows, numCols);






 // erode_kernel<<<gridSize,blockSize>>>(d_thresImage, d_erodedImage, numRows, numCols);
 // // hipDeviceSynchronize();
 //  dilate_kernel<<<gridSize,blockSize>>>(d_erodedImage, d_dilatedImage,numRows, numCols);
  
  //checkCudaErrors(hipGetLastError());
}


